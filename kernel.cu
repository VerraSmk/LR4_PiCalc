#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <stdio.h>
#include <ctime>
#include <vector>


using namespace std;

clock_t c_start, c_end;
int n = 1024 * 1024 * 32;

__global__ void count_pi(float* dev_randX, float* dev_randY, int* dev_threads_num, int n) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	int cont = 0;
	for (int i = tid * 128; i < 128 * (tid + 1); i++) {
		if (dev_randX[i] * dev_randX[i] + dev_randY[i] * dev_randY[i] < 1.0f) {
			cont++;
		}
	}
	dev_threads_num[tid] = cont;
}

int main() {

	vector<float> randX(n);
	vector<float> randY(n);

	srand((unsigned)time(NULL));
	for (int i = 0; i < n; i++) {
		randX[i] = float(rand()) / RAND_MAX;
		randY[i] = float(rand()) / RAND_MAX;
	}

	c_start = clock();
	int c_count = 0;

	for (int i = 0; i < n; i++) {
		if (randX[i] * randX[i] + randY[i] * randY[i] < 1.0f) {
			c_count++;
		}
	}
	c_end = clock();
	float t_cpu = (float)(c_end - c_start) / CLOCKS_PER_SEC;
	float c_num = float(c_count) * 4.0 / n;
	cout << "CPU Time" << endl;
	cout << c_num << endl;
	cout << "time= " << t_cpu * 1000 << " ms" << endl;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	size_t size = n * sizeof(float);
	float* dev_randX;
	float* dev_randY;
	hipMalloc((void**)&dev_randX, size);
	hipMalloc((void**)&dev_randY, size);

	hipMemcpy(dev_randX, &randX.front(), size, hipMemcpyHostToDevice);
	hipMemcpy(dev_randY, &randY.front(), size, hipMemcpyHostToDevice);

	int threadsPerBlock = 512;
	int block_num = n / (128 * threadsPerBlock);
	int* dev_threads_num;
	hipMalloc((void**)&dev_threads_num, n / 128 * sizeof(int));

	count_pi << <block_num, threadsPerBlock >> > (dev_randX, dev_randY, dev_threads_num, n);

	int* threads_num = new int[n / 128];
	hipMemcpy(threads_num, dev_threads_num, n / 128 * sizeof(int), hipMemcpyDeviceToHost);

	int g_count = 0;
	for (int i = 0; i < n / 128; i++) {
		g_count += threads_num[i];
	};

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float t_gpu1;
	hipEventElapsedTime(&t_gpu1, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	float g_num = float(g_count) * 4.0 / n;
	cout << "GPU_1 Time" << endl;
	cout << g_num << endl;
	cout << "time = " << t_gpu1 << " ms" << endl;
}